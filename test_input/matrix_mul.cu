
#include <hip/hip_runtime.h>
#include <stdio.h>

// Kernel for matrix multiplication; assumes square matrices
__global__ void matrixMul(float* A, float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; // Row index
    int col = blockIdx.x * blockDim.x + threadIdx.x; // Column index

    if(row < N && col < N) {
        float sum = 0.0f;
        for(int k = 0; k < N; k++) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

int main() {
    int N = 65536; // Matrix size (N x N)
    size_t size = N * N * sizeof(float);
    float *A, *B, *C; // Host matrices
    float *d_A, *d_B, *d_C; // Device matrices

    // Allocate host memory
    A = (float*)malloc(size);
    B = (float*)malloc(size);
    C = (float*)malloc(size);

    // Initialize matrices A and B on the host
    for(int i = 0; i < N*N; i++) {
        A[i] = 1.0; // Simplified initialization
        B[i] = 2.0;
    }

    // Allocate device memory
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy host memory to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Launch the kernel with a single thread per block for maximum inefficiency
    dim3 threadsPerBlock(1, 1);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                       (N + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matrixMul<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Clean up
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(A);
    free(B);
    free(C);

    return 0;
}
