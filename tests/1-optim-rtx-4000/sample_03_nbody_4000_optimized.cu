#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define G 6.67430e-11  // Gravitational constant

struct Body {
    float x, y, z;    // Position
    float vx, vy, vz; // Velocity
    float mass;
};

__global__ void updateVelocities(Body *bodies, int n);

int main() {
    const int numBodies = 1024;
    Body *h_bodies = new Body[numBodies];
    Body *d_bodies;

    for (int i = 0; i < numBodies; i++) {
        h_bodies[i].x = rand() % 1000;
        h_bodies[i].y = rand() % 1000;
        h_bodies[i].z = rand() % 1000;
        h_bodies[i].vx = 0;
        h_bodies[i].vy = 0;
        h_bodies[i].vz = 0;
        h_bodies[i].mass = rand() % 1000 + 100;
    }

    hipMalloc(&d_bodies, numBodies * sizeof(Body));
    hipMemcpy(d_bodies, h_bodies, numBodies * sizeof(Body), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int threadsPerBlock = 256;
    int blocksPerGrid = (numBodies + threadsPerBlock - 1) / threadsPerBlock;

    hipEventRecord(start, 0);

    updateVelocities<<<blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(Body)>>>(d_bodies, numBodies);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Elapsed time: %f ms\n", milliseconds);

    hipMemcpy(h_bodies, d_bodies, numBodies * sizeof(Body), hipMemcpyDeviceToHost);

    hipFree(d_bodies);
    delete[] h_bodies;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}

__global__ void updateVelocities(Body *bodies, int n) {
    extern __shared__ Body sharedBodies[];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int tileWidth = blockDim.x;
    float fx = 0, fy = 0, fz = 0;

    if (i < n) {
        Body myBody = bodies[i];
        for (int tile = 0; tile < gridDim.x; ++tile) {
            int idx = tile * tileWidth + threadIdx.x;
            if (idx < n) {
                sharedBodies[threadIdx.x] = bodies[idx];
            }
            __syncthreads();

            for (int j = 0; j < tileWidth; ++j) {
                if (tile * tileWidth + j < n && i != tile * tileWidth + j) {
                    float dx = sharedBodies[j].x - myBody.x;
                    float dy = sharedBodies[j].y - myBody.y;
                    float dz = sharedBodies[j].z - myBody.z;
                    float distSqr = dx * dx + dy * dy + dz * dz + 1e-10f;
                    float invDist = rsqrtf(distSqr);
                    float invDist3 = invDist * invDist * invDist;
                    float force = G * myBody.mass * sharedBodies[j].mass * invDist3;
                    fx += force * dx;
                    fy += force * dy;
                    fz += force * dz;
                }
            }
            __syncthreads();
        }

        bodies[i].vx += fx / myBody.mass;
        bodies[i].vy += fy / myBody.mass;
        bodies[i].vz += fz / myBody.mass;
    }
}