#include <stdio.h>
#include <hip/hip_runtime.h>

// Optimized kernel for matrix multiplication using shared memory and loop unrolling
__global__ void matrixMul(int *A, int *B, int *C, int width) {
    // Define block size and tile size
    const int BLOCK_SIZE = 16;
    __shared__ int s_A[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int s_B[BLOCK_SIZE][BLOCK_SIZE];

    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;
    int row = by * BLOCK_SIZE + ty;
    int col = bx * BLOCK_SIZE + tx;

    int sum = 0;

    // Loop over tiles of the input matrices
    for (int m = 0; m < (width / BLOCK_SIZE); ++m) {
        // Load tiles into shared memory
        s_A[ty][tx] = A[row * width + (m * BLOCK_SIZE + tx)];
        s_B[ty][tx] = B[(m * BLOCK_SIZE + ty) * width + col];
        __syncthreads();

        // Compute product for the tile
        #pragma unroll
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            sum += s_A[ty][k] * s_B[k][tx];
        }
        __syncthreads();
    }

    // Write the block's results to the output matrix
    if (row < width && col < width) {
        C[row * width + col] = sum;
    }
}

// Main function to set up matrices and call the kernel
int main() {
    int width = 32; // Small size for simplicity
    int size = width * width;
    int bytes = size * sizeof(int);
    int *h_A, *h_B, *h_C;
    int *d_A, *d_B, *d_C;

    // Allocate memory on the host
    h_A = (int*)malloc(bytes);
    h_B = (int*)malloc(bytes);
    h_C = (int*)malloc(bytes);

    // Initialize matrices on the host
    for(int i = 0; i < size; i++) {
        h_A[i] = rand() % 1024;
        h_B[i] = rand() % 1024;
        h_C[i] = 0;
    }

    // Allocate memory on the device
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    // Create events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Execute the kernel
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((width + 15) / 16, (width + 15) / 16);

    // Record the start event
    hipEventRecord(start, 0);

    matrixMul<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, width);

    // Record the stop event
    hipEventRecord(stop, 0);

    // Wait for the stop event to complete
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Elapsed time: %f ms\n", milliseconds);

    // Copy result back to host
    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

    // Cleanup
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}