#include "hip/hip_runtime.h"
// source: https://github.com/emandere/CudaProject/blob/master/cudamatrix.cu

#include<ctime>
#include<iostream>
using namespace std;

#define BLOCK_SIZE 32

__global__ void gpuMM(float *A, float *B, float *C, int N)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float sum = 0.f;
        for (int n = 0; n < N; ++n)
            sum += A[row * N + n] * B[n * N + col];
        C[row * N + col] = sum;
    }

int testmatrix(int K)
{
	// Perform matrix multiplication C = A*B
	// where A, B and C are NxN matrices
	// Restricted to matrices where N = K*BLOCK_SIZE;
	int N;
	N = K*BLOCK_SIZE;
	
	//cout << "Executing Matrix Multiplcation" << endl;
	//cout << "Matrix size: " << N << "x" << N << endl;

	// Allocate memory on the host
	float *hA,*hB,*hC;
	hA = new float[N*N];
	hB = new float[N*N];
	hC = new float[N*N];

	// Initialize matrices on the host
	for (int j=0; j<N; j++){
	    for (int i=0; i<N; i++){
	    	hA[j*N+i] = 1.0f;//2.f*(j+i);
			hB[j*N+i] = 1.0f;//1.f*(j-i);
	    }
	}

	// Allocate memory on the device
	long size = N*N*sizeof(float);	// Size of the memory in bytes
	float *dA,*dB,*dC;
	hipMalloc(&dA,size);
	hipMalloc(&dB,size);
	hipMalloc(&dC,size);

	dim3 threadBlock(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid(K,K);
	
	// Copy matrices from the host to device
	hipMemcpy(dA,hA,size,hipMemcpyHostToDevice);
	hipMemcpy(dB,hB,size,hipMemcpyHostToDevice);
	
	//Execute the matrix multiplication kernel
	
	gpuMM<<<grid,threadBlock>>>(dA,dB,dC,N);
		
	
	// Allocate memory to store the GPU answer on the host
	float *C;
	C = new float[N*N];
	
	// Now copy the GPU result back to CPU
	hipMemcpy(C,dC,size,hipMemcpyDeviceToHost);
	hipFree( dA );
	hipFree( dB );
	cout<<"N "<<N<<" C[0][0] "<<C[0]<<endl;
	
	
}



int main()
{
	const int matrix_size = 5000;

	clock_t start;
	double duration;

	for (int i = 140; i < 150 ; i++)
	{
		start = std::clock();
		testmatrix(i);
	    duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
		cout <<i<< " " << duration <<"s"<< '\n';
	}
	
	return 0;
}
