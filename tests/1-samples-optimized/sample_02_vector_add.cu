#include <stdio.h>
#include <hip/hip_runtime.h>

// Optimized kernel for adding two vectors using loop unrolling and increased parallelism
__global__ void vectorAdd(int *A, int *B, int *C, int numElements) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Using grid-stride loop to handle any size of data
    for (int index = i; index < numElements; index += stride) {
        C[index] = A[index] + B[index];
    }
}

// Main function to set up and execute vector addition
int main() {
    int numElements = 1024; // Number of elements in each vector
    int size = numElements * sizeof(int);
    int *h_A, *h_B, *h_C;
    int *d_A, *d_B, *d_C;

    // Allocate memory on the host
    h_A = (int*)malloc(size);
    h_B = (int*)malloc(size);
    h_C = (int*)malloc(size);

    // Initialize input vectors
    for(int i = 0; i < numElements; i++) {
        h_A[i] = rand() % 100;
        h_B[i] = rand() % 100;
    }

    // Allocate vectors in device memory
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Launch the Vector Addition CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;

    // Record the start event
    hipEventRecord(start, 0);

    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

    // Record the stop event
    hipEventRecord(stop, 0);

    // Copy the result from device memory to host memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Wait for the stop event to complete
    hipEventSynchronize(stop);

    // Calculate and print the elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Elapsed time: %f ms\n", milliseconds);

    // Free device and host memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}